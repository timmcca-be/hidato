#include "timer.h"

#include <chrono>
#include "util.h"


GpuTimer::GpuTimer() {
      hipEventCreate(&startEvent);
      hipEventCreate(&stopEvent);
}

GpuTimer::~GpuTimer() {
      hipEventDestroy(startEvent);
      hipEventDestroy(stopEvent);
}

void GpuTimer::start() {
      hipEventRecord(startEvent, 0);
}

void GpuTimer::stop() {
      hipEventRecord(stopEvent, 0);
}

float GpuTimer::elapsed() {
      float elapsedTime;
      hipEventSynchronize(stopEvent);
      hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
      return elapsedTime;
}

void CpuTimer::start() {
      startTime = std::chrono::high_resolution_clock::now();
}

void CpuTimer::stop() {
      stopTime = std::chrono::high_resolution_clock::now();
}

float CpuTimer::elapsed() {
      return std::chrono::duration_cast<std::chrono::nanoseconds>(stopTime - startTime).count() / 1000000.0f;
}
